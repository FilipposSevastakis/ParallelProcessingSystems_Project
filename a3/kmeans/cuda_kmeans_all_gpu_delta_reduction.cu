#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	return 0; /* TODO: Copy from full-offload */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numCoords][numObjs]
                    double *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

    /* TODO: Copy from full-offload */

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta)
{
    extern __shared__ double shmem_total[]; 
    double* shmemClusters = shmem_total;
    double* delta_reduce_buff = shmem_total + numClusters*numCoords;
	/* TODO: copy me from shared version... */

	/* Get the global ID of the thread. */
    int tid = get_tid(); 

	/* TODO: copy me from shared version... */
    if (1) {

		/* TODO: copy me from shared version... */

		/* TODO: Replacing (*devdelta)+= 1.0; with reduction: 
			- each thread updates the single element of delta_reduce_buff 
			corresponding to its local id (threadIdx.x) -> 1.0 if membership changes, otherwise 0. 
			- Then, ensuring delta_reduce_buff is fully updated, its containts must be summed in delta_reduce_buff[0]
			either by one thread (lower perf) or with a tree-based reduction (similar to dot reduction example in slides) 
			- Finally, delta_reduce_buff[0] (local value in block) must be added to devdelta (global delta value), ensuring write dependencies! 
		*/    
    	/* TODO: additional steps for calculating new centroids in GPU? */
    }
    
}

__global__ static
void update_centroids(int numCoords,
                          int numClusters,
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters)    //  [numCoords][numClusters])
{
    /* TODO: Copy from full-offload */
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	double *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	double   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						double * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0; 
	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    /* TODO: Copy me from transpose version*/
    double  **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]

    printf("\n|-----------Full-offload Delta Reduction GPU Kmeans------------|\n\n");
    
    /* TODO: Copy me from transpose version*/
	for(;;);
    
    double *deviceObjects;
    double *deviceClusters, *devicenewClusters;
    int *deviceMembership;
    int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = -1; /* TODO: Calculate Grid size, e.g. number of blocks. */

	/*	Define the shared memory needed per block. 
		- BEWARE: Also add extra shmem for delta buffer.
    	- BEWARE: We can overrun our shared memory here if there are too many
    	clusters or too many coordinates! 
    	- This can lead to occupancy problems or even inability to run. 
    	- Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
    const unsigned int clusterBlockSharedDataSize = -1;

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
    }
           
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&devicenewClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&devicenewClusterSize, numClusters*sizeof(int)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
    
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemset(devicenewClusterSize, 0, numClusters*sizeof(int)));
    free(dimObjects[0]);        
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();  
    
    do {
		timing_internal = wtime(); 
		checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          
		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
		/* TODO: change invocation if extra parameters needed 
		find_nearest_cluster
			<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
			(numCoords, numObjs, numClusters,
			 deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
		*/

		hipDeviceSynchronize(); checkLastCudaError();
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);

		/* TODO: Copy dev_delta_ptr to &delta
		checkCuda(hipMemcpy(...)); */

		const unsigned int update_centroids_block_sz = (numCoords* numClusters > blockSize) ? blockSize: numCoords* numClusters;  /* TODO: can use different blocksize here if deemed better */
		const unsigned int update_centroids_dim_sz =  -1; /* TODO: calculate dim for "update_centroids" and fire it 
		update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
			(numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);  */  
		hipDeviceSynchronize(); checkLastCudaError();   
				       
		delta /= numObjs;
		//printf("delta is %f - ", delta);
		loop++; 
		//printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
    
                 	
    checkCuda(hipMemcpy(membership, deviceMembership,
                 numObjs*sizeof(int), hipMemcpyDeviceToHost));     
    checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                 numClusters*numCoords*sizeof(double), hipMemcpyDeviceToHost));  
                                   
	for (i=0; i<numClusters; i++) {
	    //if (newClusterSize[i] > 0) {
		for (j=0; j<numCoords; j++) {
		    clusters[i*numCoords + j] = dimClusters[j][i];
		}
	    //}
	}
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU_Delta_Reduction", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(devicenewClusters));
    checkCuda(hipFree(devicenewClusterSize));
    checkCuda(hipFree(deviceMembership));

    return;
}


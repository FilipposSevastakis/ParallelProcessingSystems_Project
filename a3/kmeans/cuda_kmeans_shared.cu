#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	return 0; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numCoords][numObjs]
                    double *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

	/* TODO: Copy me from transpose version*/

    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta)
{
    extern __shared__ double shmemClusters[];

	/* TODO: Copy deviceClusters to shmemClusters so they can be accessed faster. 
		BEWARE: Make sure operations is complete before any thread continues... */

	/* Get the global ID of the thread. */
    int tid = get_tid(); 

	/* TODO: Maybe something is missing here... should all threads run this? */
    if (1) {
        int   index, i;
        double dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/

        for (i=1; i<numClusters; i++) {
            /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
 
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (deviceMembership[tid] != index) {
        	/* TODO: Maybe something is missing here... is this write safe? */
            (*devdelta)+= 1.0;
        }

        /* assign the deviceMembership to object objectId */
        deviceMembership[tid] = index;
    }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	double *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	double   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						double * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    /* TODO: Copy me from transpose version*/
    double  **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
    
    double *deviceObjects;
    double *deviceClusters;
    int *deviceMembership;

    printf("\n|-----------Shared GPU Kmeans------------|\n\n");
    
    /* TODO: Copy me from transpose version*/
	for(;;);

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL); 
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime();  
    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = -1; /* TODO: Calculate Grid size, e.g. number of blocks. */

	/*	Define the shared memory needed per block.
    	- BEWARE: We can overrun our shared memory here if there are too many
    	clusters or too many coordinates! 
    	- This can lead to occupancy problems or even inability to run. 
    	- Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
    const unsigned int clusterBlockSharedDataSize = -1; 

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
    }
           
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
    
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();  
    
    do {
    	timing_internal = wtime(); 

		/* GPU part: calculate new memberships */
		        
        /* TODO: Copy clusters to deviceClusters
        checkCuda(hipMemcpy(...)); */
        
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          

		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
		
		/* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
    
    	/* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */

		/* CPU part: Update cluster centers*/
  		
        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i*numCoords + j];
        }
 
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        //printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
    
    /*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */ 
	for (;;);
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "Shmem", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}


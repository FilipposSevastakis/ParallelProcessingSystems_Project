#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

int _debug;
#include "alloc.h"
#include "kmeans.h"
#include "error.h"

/// This is the validation eps that will be used for result comparisson. 
/// If set too low implementations with very divergent double operation order might lead to errors!
/// Always check error difference in these cases and adjust eps accordingly! 
#ifdef VALIDATE
    double validation_eps = 1e-2; 
#endif 

static void usage(char *argv0) {
    char *help =
        "Usage: %s [switches]\n"
        "       -c num_clusters    : number of clusters (must be > 1)\n"
        "       -s size            : size of examined dataset\n"
        "       -n num_coords      : number of coordinates\n"
        "       -t threshold       : threshold value (default : 0.001)\n"
        "       -l loop_threshold  : iterations threshold (default : 10)\n"
        "       -d                 : enable debug mode\n"
        "       -h                 : print this help information\n";
        "GPU extras:\n";
        "       -b                 : blocksize\n";
    fprintf(stderr, help, argv0);
    exit(-1);
}

int main(int argc, char **argv)
{
    long i, j, opt;
    extern char* optarg;
    extern int optind;

	int block_size = 0; 
	
    long    numClusters=0, numCoords=0, numObjs=0;
    int   * membership;    // [numObjs]
    double * objects;       // [numObjs * numCoords] data  objects
    double * clusters;      // [numClusters * numCoords] cluster center
    double   dataset_size = 0, threshold;
    long    loop_threshold;
    double  io_timing_read;

    /* some default values */
    _debug         = 0;
    threshold      = 0.001;
    loop_threshold = 10;
    numClusters    = 0;

    printf("\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n");

    while ( (opt = getopt(argc,argv,"b:n:t:l:c:s:dh")) != EOF) {
        switch (opt) {
            case 'b': block_size = atol(optarg);
                      break;
            case 'c': numClusters = atol(optarg);
                      break;
            case 't': threshold=atof(optarg);
                      break;
            case 'l': loop_threshold=atol(optarg);
                      break;
            case 's': dataset_size=atof(optarg);
                      break;
            case 'n': numCoords=atol(optarg);
                      break;
            case 'd': _debug = 1;
                      break;
            case 'h':
            default: usage(argv[0]);
                      break;
        }
    }
    if(!block_size) error("block_size not provided for GPU version, terminating\n"); 
    
    if (numClusters <= 1)
        usage(argv[0]);

    numObjs = (dataset_size*1024*1024) / (numCoords*sizeof(double));

    if (numObjs < numClusters) {
        printf("Error: number of clusters must be larger than the number of data points to be clustered.\n");
        return 1;
    }
    printf("dataset_size = %.2f MB    numObjs = %ld    numCoords = %ld    numClusters = %ld, block_size = %d\n", dataset_size, numObjs, numCoords, numClusters, block_size);

    objects = dataset_generation(numObjs, numCoords);

    // Allocate space for clusters (coordinates of cluster centers)
    clusters = (double*)  malloc(numClusters * numCoords * sizeof(double));

#ifdef VALIDATE
    // Allocate space for validation clusters (coordinates of cluster centers)
    double* validation_clusters = (double*)  malloc(numClusters * numCoords * sizeof(double));
#endif 

    // The first numClusters elements are selected as initial centers
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++){
            clusters[i*numCoords + j] = objects[i*numCoords + j];
#ifdef VALIDATE
            validation_clusters[i*numCoords + j] = clusters[i*numCoords + j];
#endif 
		}
    // check initial cluster centers for repeatition 
    if (check_repeated_clusters(numClusters, numCoords, clusters) == 0) {
        printf("Error: some initial clusters are repeated. Please select distinct initial centers\n");
        return 1;
    }

    
    //printf("Initial cluster centers:\n");
    //for (i=0; i<numClusters; i++) {
    //    printf("clusters[%ld] =",i);
    //    for (j=0; j<numCoords; j++)
    //        printf(" %6.2f", clusters[i*numCoords + j]);
    //    printf("\n");
    //}
    

    // membership: the cluster id for each data object
    membership = (int*) malloc(numObjs * sizeof(int));

#ifdef VALIDATE
	// Perform validation run
    kmeans(objects, numCoords, numObjs, numClusters, threshold, loop_threshold, membership, validation_clusters);
#endif 
    // start the core computation
    printf("\n");
    kmeans_gpu(objects, numCoords, numObjs, numClusters, threshold, loop_threshold, membership, clusters, block_size);
    printf("\n");

    
    //printf("Final cluster centers:\n");
    //for (i=0; i<numClusters; i++) {
    //    printf("clusters[%ld] = ",i);
    //   for (j=0; j<numCoords; j++)
    //        printf("%6.2f ", clusters[i*numCoords + j]);
    //    printf("\n");
    //}
 
 #ifdef VALIDATE
 	printf("Performing validation....");
 	int ik, checked[numClusters][numCoords]; 
 	for (i=0; i<numClusters; i++) for (j=0; j<numCoords; j++) checked[i][j] = 0;
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++) if(!checked[i][j]){
            for (ik=0; ik<numClusters; ik++) if(!checked[i][j]) { 
        		    if (abs((validation_clusters[i*numCoords + j] - clusters[ik*numCoords + j])/validation_clusters[i*numCoords + j]) < validation_eps)
        		    {
        		    	checked[i][j] = 1; 
        		    	break;
        		    }
        	}
            if (!checked[i][j]) error("Validation failed: cluster[%d][%d]: %lf instead of %lf\n", i, j, clusters[i*numCoords + j],  validation_clusters[i*numCoords + j]);
           	//else printf("Validation ok: cluster[%d][%d]: %lf instead of %lf\n", i, j, clusters[ik*numCoords + j],  validation_clusters[i*numCoords + j]); 
    }
    printf("PASSED!\n");
    free(validation_clusters);
 #endif
    free(objects);
    free(membership);
    free(clusters);

    return 0;
}

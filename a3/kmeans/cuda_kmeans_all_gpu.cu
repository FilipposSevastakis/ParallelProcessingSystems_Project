#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	return 0; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numCoords][numObjs]
                    double *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */

    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
/*                          
                          TODO: If you choose to do (some of) the new centroid calculation here, you will need some extra parameters here (from "update_centroids").
*/                          
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta)
{
     extern __shared__ double shmemClusters[];

	/* TODO: copy me from shared version... */

	/* Get the global ID of the thread. */
    int tid = get_tid(); 

	/* TODO: copy me from shared version... */
    if (1) {

		/* TODO: copy me from shared version... */
    
    	/* TODO: additional steps for calculating new centroids in GPU? */
    }
}

__global__ static
void update_centroids(int numCoords,
                          int numClusters,
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters)    //  [numCoords][numClusters])
{

    /* TODO: additional steps for calculating new centroids in GPU? */
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	double *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	double   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						double * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0; 
	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    /* TODO: Copy me from transpose version*/
    double  **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]

    printf("\n|-----------Full-offload GPU Kmeans------------|\n\n");
    
    /* TODO: Copy me from transpose version*/
	for(;;);
    
    double *deviceObjects;
    double *deviceClusters, *devicenewClusters;
    int *deviceMembership;
    int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */
    
    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = -1; /* TODO: Calculate Grid size, e.g. number of blocks. */
	/*	Define the shared memory needed per block.
    	- BEWARE: We can overrun our shared memory here if there are too many
    	clusters or too many coordinates! 
    	- This can lead to occupancy problems or even inability to run. 
    	- Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
    const unsigned int clusterBlockSharedDataSize = -1; 

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
    }
           
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&devicenewClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&devicenewClusterSize, numClusters*sizeof(int)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
 
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
       
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemset(devicenewClusterSize, 0, numClusters*sizeof(int)));
    free(dimObjects[0]);
      
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   
    
    do {
        timing_internal = wtime(); 
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          
		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        /* TODO: change invocation if extra parameters needed 
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
        */

        hipDeviceSynchronize(); checkLastCudaError();
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
    
    	/* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */

     	const unsigned int update_centroids_block_sz = (numCoords* numClusters > blockSize) ? blockSize: numCoords* numClusters;  /* TODO: can use different blocksize here if deemed better */
     	const unsigned int update_centroids_dim_sz =  -1; /* TODO: calculate dim for "update_centroids" and fire it 
     	update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
            (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);  */  
        hipDeviceSynchronize(); checkLastCudaError();   
                       
        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        //printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
                  	
    checkCuda(hipMemcpy(membership, deviceMembership,
                 numObjs*sizeof(int), hipMemcpyDeviceToHost));     
    checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                 numClusters*numCoords*sizeof(double), hipMemcpyDeviceToHost));  
                                   
	for (i=0; i<numClusters; i++) {
		for (j=0; j<numCoords; j++) {
		    clusters[i*numCoords + j] = dimClusters[j][i];
		}
	}
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU", blockSize, timing/loop, timer_min, timer_max);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(devicenewClusters));
    checkCuda(hipFree(devicenewClusterSize));
    checkCuda(hipFree(deviceMembership));

    return;
}

